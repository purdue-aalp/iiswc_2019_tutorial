// This program computes the sum of two vectors on the GPU using CUDA
// By: Roland Green


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

using std::cout;
using std::endl;

// Vector Addition kernel
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  // Global threadID calculation
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Boundary check
  if (tid < N) c[tid] = a[tid] + b[tid];
}

int main() {
  // Size of our arrays
  int N = 1 << 10;
  size_t bytes = N * sizeof(int);

  // Host pointers
  int *h_a, *h_b, *h_c;
  h_a = new int[N];
  h_b = new int[N];
  h_c = new int[N];

  // Initialize data
  for (int i = 0; i < N; i++) {
    h_a[i] = rand() % 100;
    h_b[i] = rand() % 100;
  }

  // Device pointers
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  // Set CTA and Grid dimensions
  int THREADS = 1024;
  int BLOCKS = N / THREADS;

  // Launch the kernel
  vectorAdd<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, N);

  // Copy the data back
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // Functional test
  for (int i = 0; i < N; i++) {
    assert(h_c[i] == h_a[i] + h_b[i]);
  }

  cout << "COMPLETED SUCCESSFULLY!" << endl;

  return 0;
}
