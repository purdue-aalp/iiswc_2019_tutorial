
#include <hip/hip_runtime.h>
__global__ void vectorAdd(int *a, int *b, int *c){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

int main() {
	int N = 1 << 10;
	size_t bytes = N * sizeof(int);

	int *h_a, *h_b, *h_c;
	h_a = new int[N];
	h_b = new int[N];
	h_c = new int[N];
	for( int i = 0; i < N; i++ ){
		h_a[i] = 1;
		h_b[i] = 2;
	}

	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	vectorAdd<<<N / 128, 128>>>(d_a, d_b, d_c);

	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	return 0;
}
